#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "hip/hip_runtime.h"
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/core/core.hpp>
#include <opencv2/video/tracking.hpp>
#include "stdafx.h"
//#include "nppdefs.h"
//#include <npp.h>

typedef unsigned char uchar;
typedef unsigned int uint;

#define BLOCK_SIZE 32
#define CV_64FC1 double
#define CV_32F float
#define CV_8U uchar

int iDivUp(int a, int b)
{
	return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

__device__
double maxVal(double blue, double green, double red) {
	if ((blue >= green) && (blue >= red))
		return blue;
	else if ((green >= blue) && (green >= red))
		return green;
	else
		return red;
}

__device__
double minVal(double blue, double green, double red) {
	if ((blue <= green) && (blue <= red))
		return blue;
	else if ((green <= blue) && (green <= red))
		return green;
	else
		return red;
}

// Transfert img to imgout to see how opencv image can be acces in GPGPU
__global__ void Kernel_Tst_Img_CV_8U(uchar *img, uchar *imgout, int ImgWidth, int imgHeigh)
{
	int ImgNumColonne = blockIdx.x  * blockDim.x + threadIdx.x;
	int ImgNumLigne = blockIdx.y  * blockDim.y + threadIdx.y;
	int Index = (ImgNumLigne * ImgWidth + ImgNumColonne * 3);

	if ((ImgNumColonne < ImgWidth / 3) && (ImgNumLigne < imgHeigh))
	{
		/* Kernel Code Here */

		double blue = (double)img[Index] / 255;
		double green = (double)img[Index + 1] / 255;
		double red = (double)img[Index + 2] / 255;

		double cMax = maxVal(blue, green, red);

		double cMin = minVal(blue, green, red);

		double delta = cMax - cMin;

		//	HUE
		double hue = 0;
		if (blue == cMax) {
			hue = 60 * ((red - green) / delta + 4);
		}
		else if (green == cMax) {
			hue = 60 * ((blue - red) / delta + 2);
		}
		else if (red == cMax) {
			hue = 60 * ((green - blue) / delta);
			if (hue < 0)
				hue += 360;
		}

		//	SATURATION
		double saturation = 0;
		if (cMax != 0) {
			saturation = delta / cMax;
		}

		//	VALUE
		double value = cMax;

		imgout[Index] = (uchar)(hue / 2);
		imgout[Index + 1] = (uchar)(saturation * 255);
		imgout[Index + 2] = (uchar)(value * 255);
	}

	return;
}

extern "C" bool GPGPU_TstImg_CV_8U(cv::Mat* img, cv::Mat* GPGPUimg)
{
	hipError_t cudaStatus;
	uchar *devImage;
	uchar *devImageOut;

	unsigned int ImageSize = img->rows * img->step1();// step number of bytes in each row

													  // Allocate memory for image
	cudaStatus = hipMalloc((void**)&devImage, ImageSize);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	// Upload the image to the GPU
	cudaStatus = hipMemcpy(devImage, img->data, ImageSize, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	//dim3 dimGrid(iDivUp(img->step1(), BLOCK_SIZE), iDivUp(img->cols, BLOCK_SIZE));
	dim3 dimGrid(iDivUp(img->cols, BLOCK_SIZE), iDivUp(img->rows, BLOCK_SIZE));


	// Test only
	// Allocate memory for the result image 
	cudaStatus = hipMalloc((void**)&devImageOut, ImageSize);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}


	Kernel_Tst_Img_CV_8U << <dimGrid, dimBlock >> >(devImage, devImageOut, img->step1(), img->rows);
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	//Wait for the kernel to end
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipDeviceSynchronize failed!");
		goto Error;
	}

	// Download the result image from gpu
	cudaStatus = hipMemcpy(GPGPUimg->data, devImageOut, ImageSize, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(devImage);
	hipFree(devImageOut);

	return cudaStatus;
}
// Transfert img to imgout to see how opencv image can be acces in GPGPU

//	TODO add color
__global__ 
void Kernel_ThresholdHSV(uchar *img, uchar *imgout, int ImgWidth, int imgHeigh, int minHue, int maxHue, int* backGroundColor, bool replaceForeground, int* ForegroundColor)
{
	int ImgNumColonne = blockIdx.x  * blockDim.x + threadIdx.x;
	int ImgNumLigne = blockIdx.y  * blockDim.y + threadIdx.y;
	int Index = (ImgNumLigne * ImgWidth) + (ImgNumColonne * 3);

	if ((ImgNumColonne < ImgWidth / 3) && (ImgNumLigne < imgHeigh))
	{
		int hue = img[Index];
		int saturation = img[Index + 1];
		int value = img[Index + 2];

		if (hue > minHue && hue < maxHue) {	//	Background-Green is black by default
			imgout[Index] = backGroundColor[0];
			imgout[Index + 1] = backGroundColor[1];
			imgout[Index + 2] = backGroundColor[2];
		}
		else {
			//	REPLACE WITH RGB COLORS
			if (replaceForeground) {
				imgout[Index] = ForegroundColor[0];
				imgout[Index + 1] = ForegroundColor[1];
				imgout[Index + 2] = ForegroundColor[2];
			}
			else {
				imgout[Index] = img[Index];
				imgout[Index + 1] = img[Index + 1];
				imgout[Index + 2] = img[Index + 2];
			}		
		}
	}

	return;
}

int defaultForegroundColor[3] = { 255, 255, 255 };
extern "C" bool GPGPU_BackGroundSubstractionHSV(cv::Mat* imgHSV, cv::Mat* GPGPUimg, int minHue, int maxHue,
	int* backGroundColor, bool replaceForeground = false, int* ForegroundColor = defaultForegroundColor)
{
	//	1. Initialize data
	hipError_t cudaStatus;
	uchar *gDevImage;
	uchar *gDevImageOut;
	int* gBgColor;
	int* gFgColor;

	uint imageSize = imgHSV->rows * imgHSV->step1();
	uint ColorSize = sizeof(int) * 3;

	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(iDivUp(imgHSV->cols, BLOCK_SIZE), iDivUp(imgHSV->rows, BLOCK_SIZE));

	//	2. Allocation data
	cudaStatus = hipMalloc(&gDevImage, imageSize);
	cudaStatus = hipMalloc(&gDevImageOut, imageSize);
	cudaStatus = hipMalloc(&gBgColor, ColorSize);
	cudaStatus = hipMalloc(&gFgColor, ColorSize);

	//	3. Copy data on GPU
	cudaStatus = hipMemcpy(gDevImage, imgHSV->data, imageSize, hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(gBgColor, backGroundColor, ColorSize, hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(gFgColor, ForegroundColor, ColorSize, hipMemcpyHostToDevice);

	//	4. Launch kernel
	Kernel_ThresholdHSV << <dimGrid, dimBlock >> >(gDevImage, gDevImageOut, imgHSV->step1(), imgHSV->rows, minHue, maxHue, gBgColor, replaceForeground, gFgColor);	

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	//Wait for the kernel to end
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipDeviceSynchronize failed!");
		goto Error;
	}

	//	5. Copy data on CPU
	cudaStatus = hipMemcpy(GPGPUimg->data, gDevImageOut, imageSize, hipMemcpyDeviceToHost);

	//	6. Free GPU memory
Error:
	hipFree(gDevImage);
	hipFree(gDevImageOut);

	return cudaStatus;
}

__device__
int absGrad(int grad) {
	if (grad < 0) {
		return -1 * grad;
	}
	else {
		return grad;
	}
}

__global__ 
void Kernel_Sobel(uchar* img, uchar* imgout, int ImgWidth, int imgHeigh) // , int* maskX, int* maskY
{	
	int ImgNumColonne = blockIdx.x  * blockDim.x + threadIdx.x;
	int ImgNumLigne = blockIdx.y  * blockDim.y + threadIdx.y;

	int Index = (ImgNumLigne * ImgWidth) + (ImgNumColonne * 3);
	int IndexGray = (ImgNumLigne * (ImgWidth / 3)) + (ImgNumColonne);

	if ((ImgNumColonne < (ImgWidth / 3) - 2) && (ImgNumLigne < imgHeigh - 2)) {

		int i = Index;
		int gradX = img[i] * -3 + img[i + 3] * 0 + img[i + 6] * 3;
		i = ((ImgNumLigne + 1) * ImgWidth) + (ImgNumColonne * 3);
		gradX += img[i] * -10 + img[i + 3] * 0 + img[i + 6] * 10;
		i = ((ImgNumLigne + 2) * ImgWidth) + (ImgNumColonne * 3);
		gradX += img[i] * -3 + img[i + 3] * 0 + img[i + 6] * 3;

		i = Index;
		int gradY = img[i] * -3 + img[i + 3] * -10 + img[i + 6] * -3;
		i = ((ImgNumLigne + 1) * ImgWidth) + (ImgNumColonne * 3);
		gradY += img[i] * 0 + img[i + 3] * 0 + img[i + 6] * 0;
		i = ((ImgNumLigne + 2) * ImgWidth) + (ImgNumColonne * 3);
		gradY += img[i] * 3 + img[i + 3] * 10 + img[i + 6] * 3;


		int grad = absGrad(gradX) + absGrad(gradY);
		int norm = grad * 0.0625;

		imgout[IndexGray] = norm;

	}

	return;
}

extern "C" bool GPGPU_Sobel(cv::Mat* imgTresh, cv::Mat* Grayscale)
{
	//	1. Initialize data
	hipError_t cudaStatus;
	uchar* gDevImage;
	uchar* gDevImageOut;

	uint imageSize = imgTresh->rows * imgTresh->step1(); //	3x greater than gradientSize
	uint gradientSize = imgTresh->rows * imgTresh->cols * sizeof(uchar); 

	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(iDivUp(imgTresh->cols, BLOCK_SIZE), iDivUp(imgTresh->rows, BLOCK_SIZE));

	//	2. Allocation data
	cudaStatus = hipMalloc(&gDevImage, imageSize);
	cudaStatus = hipMalloc(&gDevImageOut, gradientSize);

	//	3. Copy data on GPU
	cudaStatus = hipMemcpy(gDevImage, imgTresh->data, imageSize, hipMemcpyHostToDevice);

	//	4. Launch kernel
	Kernel_Sobel << <dimGrid, dimBlock >> >(gDevImage, gDevImageOut, imgTresh->step1(), imgTresh->rows);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	//Wait for the kernel to end
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipDeviceSynchronize failed!");
		goto Error;
	}

	//	5. Copy data on CPU
	cudaStatus = hipMemcpy(Grayscale->data, gDevImageOut, gradientSize, hipMemcpyDeviceToHost);

	//	6. Free GPU memory
Error:
	hipFree(gDevImage);
	hipFree(gDevImageOut);

	return cudaStatus;
}
