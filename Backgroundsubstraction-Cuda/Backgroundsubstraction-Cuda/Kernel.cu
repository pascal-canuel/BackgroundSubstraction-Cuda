#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "hip/hip_runtime.h"
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/core/core.hpp>
#include <opencv2/video/tracking.hpp>
#include "stdafx.h"
//#include "nppdefs.h"
//#include <npp.h>

typedef unsigned char uchar;
typedef unsigned int uint;

#define BLOCK_SIZE 32
#define CV_64FC1 double
#define CV_32F float
#define CV_8U uchar

int iDivUp(int a, int b)
{
	return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

__device__
double maxVal(double blue, double green, double red) {
	if ((blue >= green) && (blue >= red))
		return blue;
	else if ((green >= blue) && (green >= red))
		return green;
	else
		return red;
}

__device__
double minVal(double blue, double green, double red) {
	if ((blue <= green) && (blue <= red))
		return blue;
	else if ((green <= blue) && (green <= red))
		return green;
	else
		return red;
}

// Transfert img to imgout to see how opencv image can be acces in GPGPU
__global__ void Kernel_Tst_Img_CV_8U(uchar *img, uchar *imgout, int ImgWidth, int imgHeigh)
{
	int ImgNumColonne = blockIdx.x  * blockDim.x + threadIdx.x;
	int ImgNumLigne = blockIdx.y  * blockDim.y + threadIdx.y;
	int Index = (ImgNumLigne * ImgWidth + ImgNumColonne * 3);

	if ((ImgNumColonne < ImgWidth / 3) && (ImgNumLigne < imgHeigh))
	{
		/* Kernel Code Here */

		double blue = (double)img[Index] / 255;
		double green = (double)img[Index + 1] / 255;
		double red = (double)img[Index + 2] / 255;

		double cMax = maxVal(blue, green, red);

		double cMin = minVal(blue, green, red);

		double delta = cMax - cMin;

		//	HUE
		double hue = 0;
		if (blue == cMax) {
			hue = 60 * ((red - green) / delta + 4);
		}
		else if (green == cMax) {
			hue = 60 * ((blue - red) / delta + 2);
		}
		else if (red == cMax) {
			hue = 60 * ((green - blue) / delta);
			if (hue < 0)
				hue += 360;
		}

		//	SATURATION
		double saturation = 0;
		if (cMax != 0) {
			saturation = delta / cMax;
		}

		//	VALUE
		double value = cMax;

		imgout[Index] = (uchar)(hue / 2);
		imgout[Index + 1] = (uchar)(saturation * 255);
		imgout[Index + 2] = (uchar)(value * 255);
	}

	return;
}

extern "C" bool GPGPU_TstImg_CV_8U(cv::Mat* img, cv::Mat* GPGPUimg)
{
	hipError_t cudaStatus;
	uchar *devImage;
	uchar *devImageOut;

	unsigned int ImageSize = img->rows * img->step1();// step number of bytes in each row

													  // Allocate memory for image
	cudaStatus = hipMalloc((void**)&devImage, ImageSize);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	// Upload the image to the GPU
	cudaStatus = hipMemcpy(devImage, img->data, ImageSize, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	//dim3 dimGrid(iDivUp(img->step1(), BLOCK_SIZE), iDivUp(img->cols, BLOCK_SIZE));
	dim3 dimGrid(iDivUp(img->cols, BLOCK_SIZE), iDivUp(img->rows, BLOCK_SIZE));


	// Test only
	// Allocate memory for the result image 
	cudaStatus = hipMalloc((void**)&devImageOut, ImageSize);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}


	Kernel_Tst_Img_CV_8U << <dimGrid, dimBlock >> >(devImage, devImageOut, img->step1(), img->rows);
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	//Wait for the kernel to end
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipDeviceSynchronize failed!");
		goto Error;
	}

	// Download the result image from gpu
	cudaStatus = hipMemcpy(GPGPUimg->data, devImageOut, ImageSize, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(devImage);
	hipFree(devImageOut);

	return cudaStatus;
}
// Transfert img to imgout to see how opencv image can be acces in GPGPU

//	TODO add color
__global__ 
void Kernel_ThresholdHSV(uchar *img, uchar *imgout, int ImgWidth, int imgHeigh, int minHue, int maxHue, int* backGroundColor, bool replaceForeground, int* ForegroundColor)
{
	int ImgNumColonne = blockIdx.x  * blockDim.x + threadIdx.x;
	int ImgNumLigne = blockIdx.y  * blockDim.y + threadIdx.y;
	int Index = (ImgNumLigne * ImgWidth) + (ImgNumColonne * 3);

	if ((ImgNumColonne < ImgWidth / 3) && (ImgNumLigne < imgHeigh))
	{
		int hue = img[Index];
		int saturation = img[Index + 1];
		int value = img[Index + 2];

		if (hue > minHue && hue < maxHue) {	//	Background-Green is black by default
			imgout[Index] = backGroundColor[0];
			imgout[Index + 1] = backGroundColor[1];
			imgout[Index + 2] = backGroundColor[2];
		}
		else {
			//	REPLACE WITH RGB COLORS
			if (replaceForeground) {
				imgout[Index] = ForegroundColor[0];
				imgout[Index + 1] = ForegroundColor[1];
				imgout[Index + 2] = ForegroundColor[2];
			}
			else {
				imgout[Index] = img[Index];
				imgout[Index + 1] = img[Index + 1];
				imgout[Index + 2] = img[Index + 2];
			}		
		}
	}

	return;
}

int defaultForegroundColor[3] = { 255, 255, 255 };
extern "C" bool GPGPU_BackGroundSubstractionHSV(cv::Mat* imgHSV, cv::Mat* GPGPUimg, int minHue, int maxHue,
	int* backGroundColor, bool replaceForeground = false, int* ForegroundColor = defaultForegroundColor)
{
	//	1. Initialize data
	hipError_t cudaStatus;
	uchar *gDevImage;
	uchar *gDevImageOut;
	int* gBgColor;
	int* gFgColor;

	uint imageSize = imgHSV->rows * imgHSV->step1();
	uint ColorSize = sizeof(int) * 3;

	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(iDivUp(imgHSV->cols, BLOCK_SIZE), iDivUp(imgHSV->rows, BLOCK_SIZE));

	//	2. Allocation data
	cudaStatus = hipMalloc(&gDevImage, imageSize);
	cudaStatus = hipMalloc(&gDevImageOut, imageSize);
	cudaStatus = hipMalloc(&gBgColor, ColorSize);
	cudaStatus = hipMalloc(&gFgColor, ColorSize);

	//	3. Copy data on GPU
	cudaStatus = hipMemcpy(gDevImage, imgHSV->data, imageSize, hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(gBgColor, backGroundColor, ColorSize, hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(gFgColor, ForegroundColor, ColorSize, hipMemcpyHostToDevice);

	//	4. Launch kernel
	Kernel_ThresholdHSV << <dimGrid, dimBlock >> >(gDevImage, gDevImageOut, imgHSV->step1(), imgHSV->rows, 38, 89, gBgColor, replaceForeground, gFgColor);	//	Green Hue range 38-98

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	//Wait for the kernel to end
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipDeviceSynchronize failed!");
		goto Error;
	}

	//	5. Copy data on CPU
	cudaStatus = hipMemcpy(GPGPUimg->data, gDevImageOut, imageSize, hipMemcpyDeviceToHost);

	//	6. Free GPU memory
Error:
	hipFree(gDevImage);
	hipFree(gDevImageOut);

	return cudaStatus;
}

__device__
int abs(int grad) {
	if (grad < 0) {
		return -1 * grad;
	}
	else {
		return grad;
	}
}

__global__ void Kernel_Sobel(uchar* img, uchar* imgout, int ImgWidth, int imgHeigh, int* gradientAll) // , int* maskX, int* maskY
{
	int ImgNumColonne = blockIdx.x  * blockDim.x + threadIdx.x;
	int ImgNumLigne = blockIdx.y  * blockDim.y + threadIdx.y;
	int Index = (ImgNumLigne * ImgWidth) + (ImgNumColonne * 3);


	int nani = (ImgNumLigne * (ImgWidth/3)) + ImgNumColonne;

	if ((ImgNumColonne < ImgWidth / 3) && (ImgNumLigne < imgHeigh)) 
	{

		if ((ImgNumColonne == ImgWidth - 1) || (ImgNumLigne == imgHeigh - 1) || (ImgNumColonne == ImgWidth - 2) || (ImgNumLigne == imgHeigh - 2)) {
			imgout[Index] = 0;
			imgout[Index + 1] = 0;
			imgout[Index + 2] = 0;

			gradientAll[nani] = 0;
		}
		else {
			int y = ImgNumLigne; // change imgnumligne pour y
			int x = ImgNumColonne;
			int i = Index;
			//imgout ->>> int 

			//	Gradient X ne pas calculer * 0
			int gradX = img[i] * -1 + img[i + 1] * 0 + img[i + 2] * 1;
			i = ((ImgNumLigne + 1) * ImgWidth) + (ImgNumColonne * 3);
			gradX += img[i] * -2 + img[i + 1] * 0 + img[i + 2] * 2;
			i = ((ImgNumLigne + 2) * ImgWidth) + (ImgNumColonne * 3);
			gradX += img[i] * -1 + img[i + 1] * 0 + img[i + 2] * 1;

			i = (ImgNumLigne * ImgWidth) + (ImgNumColonne * 3);

			//	Gradient Y
			int gradY = img[i] * -1 + img[i + 1] * -2 + img[i + 2] * -1;
			i = ((ImgNumLigne + 1) * ImgWidth) + (ImgNumColonne * 3);
			gradY += img[i] * 0 + img[i + 1] * 0 + img[i + 2] * 0;
			i = ((ImgNumLigne + 2) * ImgWidth) + (ImgNumColonne * 3);
			gradY += img[i] * 1 + img[i + 1] * 2 + img[i + 2] * 1;

			//	Gradient 
			int gradient = abs(gradX) + abs(gradY);
			int norm = gradient * 0.125;

			imgout[Index] = norm;
			imgout[Index + 1] = norm;
			imgout[Index + 2] = norm;

			
			gradientAll[nani] = norm;
		}	
	}

	return;
}

extern "C" bool GPGPU_Sobel(cv::Mat* imgTresh, cv::Mat* GPGPUimg, cv::Mat* Grayscale)
{
	//int maskX[9] = { -1, 0, 1,  -2, 0, 2,  -1, 0, 1 };
	//int maskY[9] = { -1, -2, -1,  0, 0, 0,  1, 2, 1 };

	//	1. Initialize data
	hipError_t cudaStatus;
	uchar* gDevImage;
	uchar* gDevImageOut;
	int* gGradient;
	//int* gX;
	//int* gY;

	uint imageSize = imgTresh->rows * imgTresh->step1();
	uint gradientSize = imgTresh->rows * imgTresh->cols * sizeof(int);
	//uint maskSize = sizeof(maskX); //	Could be maskY

	/// Sobel
	int* gradient = new int[imgTresh->rows * imgTresh->cols];
	///

	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(iDivUp(imgTresh->cols, BLOCK_SIZE), iDivUp(imgTresh->rows, BLOCK_SIZE));

	//	2. Allocation data
	cudaStatus = hipMalloc(&gDevImage, imageSize);
	cudaStatus = hipMalloc(&gDevImageOut, imageSize);
	cudaStatus = hipMalloc(&gGradient, gradientSize);
	//cudaStatus = hipMalloc(&gX, maskSize);
	//cudaStatus = hipMalloc(&gY, maskSize);


	//	3. Copy data on GPU
	cudaStatus = hipMemcpy(gDevImage, imgTresh->data, imageSize, hipMemcpyHostToDevice);
	//cudaStatus = hipMemcpy(gX, maskX, maskSize, hipMemcpyHostToDevice);
	//cudaStatus = hipMemcpy(gY, maskY, maskSize, hipMemcpyHostToDevice);

	//	4. Launch kernel
	Kernel_Sobel << <dimGrid, dimBlock >> >(gDevImage, gDevImageOut, imgTresh->step1(), imgTresh->rows, gGradient); // , gX, gY

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	//Wait for the kernel to end
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipDeviceSynchronize failed!");
		goto Error;
	}

	//	5. Copy data on CPU
	cudaStatus = hipMemcpy(GPGPUimg->data, gDevImageOut, imageSize, hipMemcpyDeviceToHost);
	cudaStatus = hipMemcpy(gradient, gGradient, imageSize, hipMemcpyDeviceToHost);

	//	6. Free GPU memory
Error:
	hipFree(gDevImage);
	hipFree(gDevImageOut);
	hipFree(gGradient);

	return cudaStatus;
}
